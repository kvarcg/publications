#include "hip/hip_runtime.h"

/*
 * Copyright (c) 2008 - 2009 NVIDIA Corporation.  All rights reserved.
 *
 * NVIDIA Corporation and its licensors retain all intellectual property and proprietary
 * rights in and to this software, related documentation and any modifications thereto.
 * Any use, reproduction, disclosure or distribution of this software and related
 * documentation without an express license agreement from NVIDIA Corporation is strictly
 * prohibited.
 *
 * TO THE MAXIMUM EXTENT PERMITTED BY APPLICABLE LAW, THIS SOFTWARE IS PROVIDED *AS IS*
 * AND NVIDIA AND ITS SUPPLIERS DISCLAIM ALL WARRANTIES, EITHER EXPRESS OR IMPLIED,
 * INCLUDING, BUT NOT LIMITED TO, IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A
 * PARTICULAR PURPOSE.  IN NO EVENT SHALL NVIDIA OR ITS SUPPLIERS BE LIABLE FOR ANY
 * SPECIAL, INCIDENTAL, INDIRECT, OR CONSEQUENTIAL DAMAGES WHATSOEVER (INCLUDING, WITHOUT
 * LIMITATION, DAMAGES FOR LOSS OF BUSINESS PROFITS, BUSINESS INTERRUPTION, LOSS OF
 * BUSINESS INFORMATION, OR ANY OTHER PECUNIARY LOSS) ARISING OUT OF THE USE OF OR
 * INABILITY TO USE THIS SOFTWARE, EVEN IF NVIDIA HAS BEEN ADVISED OF THE POSSIBILITY OF
 * SUCH DAMAGES
 */

#include "commonStructs.h"
#include "helpers.h"
#include "random.h"

rtBuffer<float4, 2>       output_buffer;
rtDeclareVariable(int	, max_depth, , );
rtDeclareVariable(float	, scene_epsilon, , );
rtDeclareVariable(rtObject, top_object, , );
rtDeclareVariable(Ray, ray, rtCurrentRay, );
rtDeclareVariable(uint2, launch_index, rtLaunchIndex, );

namespace camera
{
	rtDeclareVariable(float3, eye, , );
	rtDeclareVariable(float3, U, , );
	rtDeclareVariable(float3, V, , );
	rtDeclareVariable(float3, W, , );
}

namespace color
{
	rtTextureSampler<float4, 2> envmap;
	rtDeclareVariable(float3, background, , );
	rtDeclareVariable(float3, exception, , );
}

//
// Perspective Camera
//
RT_PROGRAM void pinhole_camera()
{
	unsigned int samples_per_pixel = pt::sqrt_num_samples*pt::sqrt_num_samples;

	size_t2 screen		= output_buffer.size();
	float2 inv_screen	= 1.0f / make_float2(screen) * 2.f;
	float2 pixel		= (make_float2(launch_index)) * inv_screen - 1.f;
#if defined (AA)		
	float2 jitter_scale = inv_screen / pt::sqrt_num_samples;
	unsigned int x = samples_per_pixel % pt::sqrt_num_samples;
	unsigned int y = samples_per_pixel / pt::sqrt_num_samples;
#endif

	unsigned int seed	= tea<16>(screen.x*launch_index.y + launch_index.x, pt::frame_number);
	
	float3 result = make_float3(0.0f);
	do
	{
#if defined (AA)	
		float2 jitter	= make_float2(x - rnd(seed), y - rnd(seed));
		float2 d		= pixel + jitter*jitter_scale;
#else
		float2 d		= pixel;
#endif
		float3 ray_origin	 = camera::eye;
		float3 ray_direction = normalize(d.x*camera::U + d.y*camera::V + camera::W);

		PerRayData_radiance prd;
		prd.result		 = make_float3(0.f);
		prd.attenuation  = make_float3(1.f);
		prd.done		 = false;
		prd.seed		 = seed;
		prd.depth		 = 0;

		while(!prd.done && prd.depth < max_depth)
		{
			Ray ray = make_Ray(ray_origin, ray_direction, pt::radiance_ray_type, scene_epsilon, RT_DEFAULT_MAX);
			rtTrace(top_object, ray, prd);
			
			prd.result += prd.radiance * prd.attenuation;

			// Monte Carlo strategy (Russian Roulette) as stopping criterion
			//if (prd.depth >= rr_begin_depth)
			//{
				//float pcont = fmaxf(prd.attenuation);
			//	if (rnd(prd.seed) >= pcont)
			//		break;
				//prd.attenuation /= pcont;
			//}
			
			prd.depth++;
			ray_origin	  = prd.origin;
			ray_direction = prd.direction;
		} 

		result += prd.result;
		seed	= prd.seed;
	} 
	while (--samples_per_pixel);

	float3 pixel_color = result / (pt::sqrt_num_samples*pt::sqrt_num_samples);
	if (pt::frame_number > 1)
	{
		float a = 1.0f / (float)pt::frame_number;
		float b = ((float)pt::frame_number - 1.0f) * a;
		float3 old_color = make_float3(output_buffer[launch_index]);
		output_buffer[launch_index] = make_float4(a * pixel_color + b * old_color, 0.0f);
	}
	else
		output_buffer[launch_index] = make_float4(pixel_color, 0.0f);
}
//
// Orthographic Camera
//
RT_PROGRAM void orthographic_camera()
{
	unsigned int samples_per_pixel = pt::sqrt_num_samples*pt::sqrt_num_samples;

	size_t2 screen = output_buffer.size();
	float2 inv_screen = 1.0f / make_float2(screen) * 2.f;
	float2 pixel = (make_float2(launch_index)) * inv_screen - 1.f;
#if defined (AA)	
	float2 jitter_scale = inv_screen / pt::sqrt_num_samples;
	unsigned int x = samples_per_pixel % pt::sqrt_num_samples;
	unsigned int y = samples_per_pixel / pt::sqrt_num_samples;
#endif

	unsigned int seed = tea<16>(screen.x*launch_index.y + launch_index.x, pt::frame_number);

	float3 result = make_float3(0.0f);
	do
	{
#if defined (AA)	
		float2 jitter = make_float2(x - rnd(seed), y - rnd(seed));
		float2 d = pixel + jitter*jitter_scale;
#else
		float2 d = pixel;
#endif
		float3 ray_origin = camera::eye + d.x*camera::U + d.y*camera::V;	// eye + offset in film space
		float3 ray_direction = camera::W;									// always parallel view direction

		PerRayData_radiance prd;
		prd.result = make_float3(0.f);
		prd.attenuation = make_float3(1.f);
		prd.done = false;
		prd.seed = seed;
		prd.depth = 0;

		while (!prd.done && prd.depth < max_depth)
		{
			Ray ray = make_Ray(ray_origin, ray_direction, pt::radiance_ray_type, scene_epsilon, RT_DEFAULT_MAX);
			rtTrace(top_object, ray, prd);

			prd.result += prd.radiance * prd.attenuation;

			// Monte Carlo strategy (Russian Roulette) as stopping criterion
			//if (prd.depth >= rr_begin_depth)
			//{
			//	float pcont = fmaxf(prd.attenuation);
			//	if (rnd(prd.seed) >= pcont)
			//		break;
			//	prd.attenuation /= pcont;
			//}

			prd.depth++;
			ray_origin = prd.origin;
			ray_direction = prd.direction;
		}

		result += prd.result;
		seed = prd.seed;
	} while (--samples_per_pixel);

	float3 pixel_color = result / (pt::sqrt_num_samples*pt::sqrt_num_samples);
	if (pt::frame_number > 1)
	{
		float a = 1.0f / (float)pt::frame_number;
		float b = ((float)pt::frame_number - 1.0f) * a;
		float3 old_color = make_float3(output_buffer[launch_index]);
		output_buffer[launch_index] = make_float4(a * pixel_color + b * old_color, 0.0f);
	}
	else
		output_buffer[launch_index] = make_float4(pixel_color, 0.0f);
}
//
// Returns environment map color for miss rays
//
RT_PROGRAM void envmap_miss()
{
	float theta = atan2f(ray.direction.x, ray.direction.z);
	float phi = M_PIf * 0.5f - acosf(ray.direction.y);
	float u = (theta + M_PIf) * (0.5f * M_1_PIf);
	float v = 0.5f * (1.0f + sin(phi));

	pt::prd_radiance.radiance = make_float3(tex2D(color::envmap, u, v));
	pt::prd_radiance.done = true;
}
//
// Returns background color for miss rays
//
RT_PROGRAM void background_miss()
{
	pt::prd_radiance.radiance = color::background;
	pt::prd_radiance.done = true;
}
//
// Returns solid color upon failure
//
RT_PROGRAM void exception()
{
	rtPrintf("Caught exception 0x%X at launch index (%d,%d)\n", rtGetExceptionCode(), launch_index.x, launch_index.y);
	output_buffer[launch_index] = make_float4(color::exception, 0.0f);
}
