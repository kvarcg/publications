#include "hip/hip_runtime.h"

/*
 * Copyright (c) 2008 - 2009 NVIDIA Corporation.  All rights reserved.
 *
 * NVIDIA Corporation and its licensors retain all intellectual property and proprietary
 * rights in and to this software, related documentation and any modifications thereto.
 * Any use, reproduction, disclosure or distribution of this software and related
 * documentation without an express license agreement from NVIDIA Corporation is strictly
 * prohibited.
 *
 * TO THE MAXIMUM EXTENT PERMITTED BY APPLICABLE LAW, THIS SOFTWARE IS PROVIDED *AS IS*
 * AND NVIDIA AND ITS SUPPLIERS DISCLAIM ALL WARRANTIES, EITHER EXPRESS OR IMPLIED,
 * INCLUDING, BUT NOT LIMITED TO, IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A
 * PARTICULAR PURPOSE.  IN NO EVENT SHALL NVIDIA OR ITS SUPPLIERS BE LIABLE FOR ANY
 * SPECIAL, INCIDENTAL, INDIRECT, OR CONSEQUENTIAL DAMAGES WHATSOEVER (INCLUDING, WITHOUT
 * LIMITATION, DAMAGES FOR LOSS OF BUSINESS PROFITS, BUSINESS INTERRUPTION, LOSS OF
 * BUSINESS INFORMATION, OR ANY OTHER PECUNIARY LOSS) ARISING OUT OF THE USE OF OR
 * INABILITY TO USE THIS SOFTWARE, EVEN IF NVIDIA HAS BEEN ADVISED OF THE POSSIBILITY OF
 * SUCH DAMAGES
 */

#include <optix.h>
#include <optixu/optixu_math_namespace.h>
#include "phong.h"

using namespace optix;

// Correspond to OBJ mtl params
namespace obj_material
{
	rtTextureSampler<float4, 2>   ambient_map;
	rtTextureSampler<float4, 2>   diffuse_map;		
	rtTextureSampler<float4, 2>   specular_map;		 
	
	rtDeclareVariable(int	, illum		, , );
	rtDeclareVariable(float	, phong_exp	, , );		
}

// Correspond to OBJ geom params
namespace obj_geometry
{
	rtDeclareVariable(float3, texcoord			, attribute texcoord, );
	rtDeclareVariable(float3, geometric_normal	, attribute geometric_normal, );
	rtDeclareVariable(float3, shading_normal	, attribute shading_normal, );
}

RT_PROGRAM void any_hit_shadow()
{
	path_tracingShadowed();
}

RT_PROGRAM void closest_hit_radiance()
{
	float3 direction				= ray.direction;
	float3 world_shading_normal		= normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, obj_geometry::shading_normal));
	float3 world_geometric_normal	= normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, obj_geometry::geometric_normal));
	float3 ffnormal					= faceforward(world_shading_normal, -direction, world_geometric_normal);
	float3 uv						= obj_geometry::texcoord;
	float3 black					= make_float3(0.0f, 0.0f, 0.0f);

	// grab values from textures
	// support only MTL illumination modes 0-3 (Ks is for now used as reflectivity)
	float3 Kd = make_float3(tex2D(obj_material::diffuse_map, uv.x, uv.y));
	float3 Ka = (obj_material::illum < 1) ? black : make_float3(tex2D(obj_material::ambient_map, uv.x, uv.y));
	float3 Ks = (obj_material::illum < 2) ? black : make_float3(tex2D(obj_material::specular_map, uv.x, uv.y));
	float3 Kr = (obj_material::illum < 3) ? black : Ks;

	path_tracingShade(ffnormal, Ka, Kd, Ks, Kr, obj_material::phong_exp);
}